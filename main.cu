#include <iostream>
#include <fstream>
#include <vector>
#include "cuda_error_check.h"
#include "implementation.h"
#include "utils.h"

using namespace std;

int main(int argc, char** argv){
	try {
		//declare and initialize variabls
		string usage =
		"\tCommand line arguments:\n\
                        Input file: E.g., --input in.txt\n\
                        Output path: E.g., --output out.txt\n\
                        Block size: E.g., --bsize 1024\n\
                        Block count: E.g., --bcount 2\n\
                        Method: E.g., --method 1, 2, or 3\n";
		string inputFileName;
		string outputFileName;
		ifstream inputFile;
		ofstream outputFile;
		int bsize = 0, bcount = 0;
		int method = 0;
		int device_id = 0;
		hipDeviceProp_t device_prop;
		char* device_name = NULL;

		//check that CUDA is supported and get the name of the device
		CUDAErrorCheck(hipSetDevice(device_id));
		CUDAErrorCheck(hipGetDeviceProperties(&device_prop, device_id));
		device_name = device_prop.name;
	
		//parse program arguments
		for( int i = 1; i < argc; i++ ){
			if ( !strcmp(argv[i], "--input") && i != argc-1 ) {
				inputFileName = string(argv[i+1]);
				inputFile.open(inputFileName.c_str());
			} else if( !strcmp(argv[i], "--output") && i != argc-1 ) {
				outputFileName = string(argv[i+1]);
				outputFile.open(outputFileName.c_str());
			} else if( !strcmp(argv[i], "--bsize") && i != argc-1 ) {
				bsize = atoi( argv[i+1] );
			} else if( !strcmp(argv[i], "--bcount") && i != argc-1 ) {
				bcount = atoi( argv[i+1] );
			} else if( !strcmp(argv[i], "--method") && i != argc-1 ) {
				method = atoi( argv[i+1] );
			}
		}

		//verify program arguments
		if(!inputFile){
			throw runtime_error("Failed to open specified file: " + inputFileName);
		}
		if(!outputFile){
			throw runtime_error("Failed to open specified file: " + outputFileName);
		}
		if(!inputFile.is_open() || !outputFile.is_open()){
			cerr << "Usage: " << usage;
			throw runtime_error("Initialization error happened: input/output file");
		}
		if(bsize <= 0 || bcount <= 0){
			cerr << "Usage: " << usage;
			throw runtime_error("Initialization error happened: block size/count");
		}
		if(method == 0){
			cerr << "Usage: " << usage;
			throw runtime_error("Initialization error happened: method");
		}

		//parse input file
		const char* strings; //all the strings concatenated into a single string
		int* indices; //the starting index of each string
		int totalLength; //total length of all the strings
		int numStrings; //number of strings
		parse_file(&inputFile,&strings,&indices,&totalLength,&numStrings);
		inputFile.close();

		//print program properties
		cout << "Device: " << device_name;
		cout << ", bsize: " << bsize << ", bcount: " << bcount;
		cout << ", method: " << method << endl;	
		cout << "Input file: " << inputFileName;
		cout << ", Number of strings: " << numStrings;
		cout << ", total length: " << totalLength << endl;

		//process method
		switch(method){
		case 1:
			impl1(strings, indices, totalLength, numStrings, bsize, bcount);
			break;
		case 2:
			impl2(strings, indices, totalLength, numStrings, bsize, bcount);
			break;
		case 3:
			impl3(strings, indices, totalLength, numStrings, bsize, bcount);
			break;
		default:
			cout << "Method " << method << " does not exist. Try method 1, 2, or 3.\n";
			break;
		}

		//clean program memory
		CUDAErrorCheck(hipDeviceReset());
		outputFile.close();

	} catch(const exception& e){
		cerr << e.what() << endl;
		return EXIT_FAILURE;
	} catch(...) {
		cerr << "An exception has occurred." << endl;
		return EXIT_FAILURE;
	}

	return EXIT_SUCCESS;
}
