#include "implementation.h"

void impl3(const char* strings, int* indices, int totalLength, int numStrings, int bsize, int bcount){
	Timer timer;
	char* d_strings = NULL;
	int* d_indices = NULL;

	hipMalloc((void**)&d_strings, sizeof(char)*totalLength);
	hipMalloc((void**)&indices, sizeof(int)*numStrings);

	hipMemcpy(d_strings, strings, sizeof(char)*totalLength, hipMemcpyHostToDevice);
	hipMemcpy(d_indices, indices, sizeof(int)*numStrings, hipMemcpyHostToDevice);



	timer.set();

	// put code here

	cout << "running time: " << timer.get() << " ms" << endl;
	
	// free
	hipFree(d_strings);
	hipFree(d_indices);
}
