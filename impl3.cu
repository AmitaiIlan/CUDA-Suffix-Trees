#include "implementation.h"

void impl3(char* text, int* indices, int totalLength, int numStrings, int bsize, int bcount){
	Timer timer;
	char* d_text = NULL;
	int* d_indices = NULL;

	hipMalloc((void**)&d_text, sizeof(char)*totalLength);
	hipMalloc((void**)&d_indices, sizeof(int)*numStrings);

	hipMemcpy(d_text, text, sizeof(char)*totalLength, hipMemcpyHostToDevice);
	hipMemcpy(d_indices, indices, sizeof(int)*numStrings, hipMemcpyHostToDevice);

	timer.set();

	// put code here

	cout << "running time: " << timer.get() << " ms" << endl;
	
	// free
	hipFree(d_text);
	hipFree(d_indices);
}
