#include "hip/hip_runtime.h"
#include "implementation.h"

__global__ void suffix_tree_construction(const char* text, int* indices, int totalLength, int numStrings){
	const int tid = threadIdx.x + blockDim.x*blockIdx.x;
	const int nThreads = blockDim.x*gridDim.x;
	const int iter = totalLength%nThreads == 0? totalLength/nThreads : totalLength/nThreads+1;

	for(int i = 0; i < iter; i++){
		int dataid = tid + i*nThreads;
		if(dataid < totalLength){
			
		}
	}
}

void impl1(const char* text, int* indices, int totalLength, int numStrings, int bsize, int bcount){
	Timer timer;
        char* d_text = NULL;
        int* d_indices = NULL;

        hipMalloc((void**)&d_text, sizeof(char)*totalLength);
        hipMalloc((void**)&d_indices, sizeof(int)*numStrings);

        hipMemcpy(d_text, text, sizeof(char)*totalLength, hipMemcpyHostToDevice);
        hipMemcpy(d_indices, indices, sizeof(int)*numStrings, hipMemcpyHostToDevice);

	timer.set();

	// put code here

	cout << "running time: " << timer.get() << " ms" << endl;

	// free
	hipFree(d_text);
	hipFree(d_indices);
}
