#include "hip/hip_runtime.h"
#include "implementation.h"

__global__ void constructSuffixTree(Node* root, char* text, int* indices, int totalLength, int numStrings){
	const int tid = threadIdx.x + blockDim.x*blockIdx.x;
	const int nThreads = blockDim.x*gridDim.x;
	const int iter = numStrings%nThreads == 0? numStrings/nThreads : numStrings/nThreads+1;

	for(int i = 0; i < iter; i++){
		int dataid = tid + i*nThreads;
		if(dataid < numStrings){
			int start = indices[dataid];
			int end = dataid == numStrings-1? totalLength : indices[dataid+1];
			for(; text[start] != '#'; start++){
				char c = text[start];
				char index = charToIndex(c);
				Node** address = &(root->children[index]);
				Node* child = *address;
				if(child == NULL){
					child = createNode(start,end);
					addNode(address,child,text);
				} else {
					child = createNode(start,end);
					combineNode(address,child,text);
				}
			}
		}
	}
}

__global__ void constructSuffixTree2(Node* root, char* text, int* indices, int totalLength, int start_index, int end_index){
	const int tid = threadIdx.x + blockDim.x*blockIdx.x;
	const int nThreads = blockDim.x*gridDim.x;
	int numStrings = end_index-start_index;
	const int iter = numStrings%nThreads == 0? numStrings/nThreads : numStrings/nThreads+1;

	for(int i = 0; i < iter; i++){
		int dataid = start_index + tid + i*nThreads;
		if(dataid < end_index){
			int start = indices[dataid];
			int end = dataid == numStrings-1? totalLength : indices[dataid+1];
			for(; text[start] != '#'; start++){
				char c = text[start];
				char index = charToIndex(c);
				Node** address = &(root->children[index]);
				Node* child = *address;
				if(child == NULL){
					child = createNode(start,end);
					addNode(address,child,text);
				} else {
					child = createNode(start,end);
					combineNode(address,child,text);
				}
			}
		}
	}
}

char* impl1_batches(char* text, int* indices, int totalLength, int numStrings, int bsize, int bcount){
	Timer timer;
	Node root;
	root.start=0;
	root.end=0;
	for(int i = 0; i < NUM_CHILDREN; i++)
		root.children[i] = NULL;
	root.suffixIndex = 0;

        char* d_text = NULL;
        int* d_indices = NULL;
	Node* d_root = NULL;

        hipMalloc((void**)&d_text, sizeof(char)*totalLength);
        hipMalloc((void**)&d_indices, sizeof(int)*numStrings);
        hipMalloc((void**)&d_root, sizeof(Node));

        hipMemcpy(d_text, text, sizeof(char)*totalLength, hipMemcpyHostToDevice);
        hipMemcpy(d_indices, indices, sizeof(int)*numStrings, hipMemcpyHostToDevice);
	hipMemcpy(d_root,&root,sizeof(Node),hipMemcpyHostToDevice);

	timer.set();

	int stringsPerBatch = bcount*bsize;
	int numBatches = numStrings % stringsPerBatch == 0? numStrings/stringsPerBatch : numStrings/stringsPerBatch+1;

	for(int i = 0; i < numBatches; i++){
		int start = i*stringsPerBatch;
		int end = min(start+stringsPerBatch,numStrings);
		constructSuffixTree2<<<bcount,bsize>>>(d_root,d_text,d_indices,totalLength,start,end);
		CUDAErrorCheck(hipPeekAtLastError());
		CUDAErrorCheck(hipDeviceSynchronize());
	}
	
	cout << "running time: " << timer.get() << " ms" << endl;

	//printTree<<<1,1>>>(d_root,d_text);
	//hipDeviceSynchronize();

	char* output = NULL;
	int size = getSerialSuffixTree(d_root,d_text,&output);
	printf("Output size: %d\n",size);

	// free
	hipFree(d_text);
	hipFree(d_indices);
	hipFree(d_root);

	return output;
}

char* impl1(char* text, int* indices, int totalLength, int numStrings, int bsize, int bcount){
	Timer timer;
	Node root;
	root.start=0;
	root.end=0;
	for(int i = 0; i < NUM_CHILDREN; i++)
		root.children[i] = NULL;
	root.suffixIndex = 0;

        char* d_text = NULL;
        int* d_indices = NULL;
	Node* d_root = NULL;

        hipMalloc((void**)&d_text, sizeof(char)*totalLength);
        hipMalloc((void**)&d_indices, sizeof(int)*numStrings);
        hipMalloc((void**)&d_root, sizeof(Node));

        hipMemcpy(d_text, text, sizeof(char)*totalLength, hipMemcpyHostToDevice);
        hipMemcpy(d_indices, indices, sizeof(int)*numStrings, hipMemcpyHostToDevice);
	hipMemcpy(d_root,&root,sizeof(Node),hipMemcpyHostToDevice);

	timer.set();

	constructSuffixTree<<<bcount,bsize>>>(d_root,d_text,d_indices,totalLength,numStrings);

	CUDAErrorCheck(hipPeekAtLastError());
	CUDAErrorCheck(hipDeviceSynchronize());
	
	cout << "running time: " << timer.get() << " ms" << endl;

	//printTree<<<1,1>>>(d_root,d_text);
	//hipDeviceSynchronize();

	char* output = NULL;
	int size = getSerialSuffixTree(d_root,d_text,&output);
	printf("Output size: %d\n",size);

	// free
	hipFree(d_text);
	hipFree(d_indices);
	hipFree(d_root);

	return output;
}

/*
char* impl1_batches(char** text, int** indices, int* totalLength, int* numStrings, int numBatches, int bsize, int bcount){
	Timer timer;
	Node root;
	root.start=0;
	root.end=0;
	for(int i = 0; i < NUM_CHILDREN; i++)
		root.children[i] = NULL;
	root.suffixIndex = 0;

	Node* d_root = NULL;
        hipMalloc((void**)&d_root, sizeof(Node));
	hipMemcpy(d_root,&root,sizeof(Node),hipMemcpyHostToDevice);
   
	char** d_text = NULL;
	int** d_indices = NULL;

	for(int i = 0; i < numBatches; i++){
        	hipMalloc((void**)&(d_text[i]), sizeof(char)*totalLength[i]);
        	hipMalloc((void**)&(d_indices[i]), sizeof(int)*numStrings[i]);

        	hipMemcpy(d_text[i], text, sizeof(char)*totalLength[i], hipMemcpyHostToDevice);
        	hipMemcpy(d_indices[i], indices, sizeof(int)*numStrings[i], hipMemcpyHostToDevice);
	}

	timer.set();

	for(int i = 0; i < numBatches; i++){
		constructSuffixTree<<<bcount,bsize>>>(d_root,d_text[i],d_indices[i],totalLength[i],numStrings[i]);
		CUDAErrorCheck(hipPeekAtLastError());
		CUDAErrorCheck(hipDeviceSynchronize());
	}
	
	cout << "running time: " << timer.get() << " ms" << endl;

	//printTree<<<1,1>>>(d_root,d_text);
	//hipDeviceSynchronize();

	char* output = NULL;
	int size = getSerialSuffixTree(d_root,d_text,&output);
	printf("Output size: %d\n",size);

	// free
	hipFree(d_text);
	hipFree(d_indices);
	hipFree(d_root);

	return output;
}*/
